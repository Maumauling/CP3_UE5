#include "hip/hip_runtime.h"
/*********************************************************************
run-cg.cu

Hauptprogramm. Testet Reduktion und ruft cg auf.

**********************************************************************/
#define MAIN_PROGRAM

#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include "global.h"
#include "geometry.h"
#include "linalg.h"
#include "cg.h"

int main(int argc, char **argv)
{
   printf("%s Starting...\n", argv[0]);

   int nBytes, status, N;
   double *w, *v, *x, *s, *vo, *rnormalt, *svo, *r, *ro, *rnorm, *sv, *vro, *vr, *vso, *vs;
   double iStart, iElaps;

   N=32;
   int dimx = 256;
   int dimy = 1;
   if (argc>1)
   {
      N=atoi(argv[1]);
   }
   if (argc>3)
   {
      dimx=atoi(argv[2]);
      dimy=atoi(argv[3]);
   }

   // set up device
   int dev = 0;
   hipDeviceProp_t deviceProp;
   CHECK(hipGetDeviceProperties(&deviceProp, dev));
   printf("Using Device %d: %s\n", dev, deviceProp.name);
   CHECK(hipSetDevice(dev));

   // Globale Variablen setzen:
   // Anzahl der Inneren Punkte in x- und y-Richtung
   Nx=N;
   Ny=N;
   // Gesamtanzahl der Gitterpunkte
   npts=(Nx+2)*(Ny+2);
   // Aktive Punkte - Array
   active_pts();

   // Speicherbedarf pro Vektor in Byte
   nBytes=npts*sizeof(double);

   // Speicher für Vektoren allozieren + GPU ERWEITERUNGEN
   w=(double*)malloc(nBytes);
   v=(double*)malloc(nBytes);
   
   x=(double*)malloc(nBytes);
   s=(double*)malloc(nBytes);
   vo=(double*)malloc(nBytes);
   rnormalt=(double*)malloc(nBytes);
   svo=(double*)malloc(nBytes);
   r=(double*)malloc(nBytes);
   ro=(double*)malloc(nBytes);
   rnorm=(double*)malloc(nBytes);
   sv=(double*)malloc(nBytes);
   vro=(double*)malloc(nBytes);
   vr=(double*)malloc(nBytes);
   vs=(double*)malloc(nBytes);
   vso=(double*)malloc(nBytes);
   

   // auf Null setzen + GPU ERWEITERUNGEN
   memset(w, 0, nBytes);
   memset(v, 0, nBytes);
   
   memset(x, 0, nBytes);
   memset(s, 0, nBytes);
   memset(vo, 0, nBytes);
   memset(rnormalt, 0,nBytes);
   memset(svo, 0, nBytes);
   memset(r, 0, nBytes);
   memset(ro, 0, nBytes);
   memset(rnorm, 0, nBytes);
   memset(sv, 0, nBytes);
   memset(vro, 0, nBytes);
   memset(vr, 0, nBytes);
   memset(vso, 0, nBytes);
   memset(vs, 0, nBytes);

   // Aktive Punkte ausgeben
   if ((Nx<=16)&&(Ny<=16))
      print_active();

   random_vector(w);
   random_vector(v);
   double *d_v, *d_w, *d_x, *d_s, *d_vo, *d_rnorm_alt, *d_svo, *d_r, *d_ro, *d_rnorm, *d_sv, *d_vro, *d_vr, *d_vso, *d_vs;
   CHECK(hipMalloc((void **)&d_v, nBytes));
   CHECK(hipMalloc((void **)&d_w, nBytes));
   
   CHECK(hipMalloc((void **)&d_x, nBytes)); // GPU ERWEITERUNGEN
   CHECK(hipMalloc((void **)&d_s, nBytes));
   CHECK(hipMalloc((void **)&d_vo, nBytes));
   CHECK(hipMalloc((void **)&d_rnorm_alt, nBytes));
   CHECK(hipMalloc((void **)&d_svo, nBytes));
   CHECK(hipMalloc((void **)&d_r, nBytes));
   CHECK(hipMalloc((void **)&d_ro, nBytes));
   CHECK(hipMalloc((void **)&d_rnorm, nBytes));
   CHECK(hipMalloc((void **)&d_sv, nBytes));
   CHECK(hipMalloc((void **)&d_vro, nBytes));
   CHECK(hipMalloc((void **)&d_vr, nBytes));
   CHECK(hipMalloc((void **)&d_vso, nBytes));
   CHECK(hipMalloc((void **)&d_vs, nBytes));
   
   // transfer data from host to device
   CHECK(hipMemcpy(d_v, v, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_w, w, nBytes, hipMemcpyHostToDevice));
   
   CHECK(hipMemcpy(d_x, x, nBytes, hipMemcpyHostToDevice)); // GPU ERWEITERUNGEN
   CHECK(hipMemcpy(d_s, s, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_vo, vo, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_rnorm_alt, rnormalt, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_svo, svo, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_r, r, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_ro, ro, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_rnorm, rnorm, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_sv, sv, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_vro, vro, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_vr, vr, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_vso, vso, nBytes, hipMemcpyHostToDevice));
   CHECK(hipMemcpy(d_vs, vs, nBytes, hipMemcpyHostToDevice));
   
   // invoke kernel at host side
   block.x=dimx;
   block.y=dimy;
   block.z=1;
   grid.x=(Nx + block.x - 1) / block.x;
   grid.y=(Ny + block.y - 1) / block.y;
   grid.z=1;

   // Test reduction
   /*int Nunroll=8;
   if (npts>256 && Nunroll>1)
   {
      double cpu_sum=0.0;
      iStart = seconds();
      for (int i = 0; i < npts; i++) cpu_sum += v[i];
      iElaps = seconds() - iStart;
      printf("cpu reduce      elapsed %f sec cpu_sum: %f\n", iElaps, cpu_sum);

      dim3 block2 (256,1);
      int nblk = (npts + (block2.x*Nunroll) - 1)/(block2.x*Nunroll);
      dim3 grid2 (nblk,1);
      CHECK(hipMalloc((void **)&d_x, nblk*sizeof(double)));
      CHECK(hipMemset(d_x,0,nblk*sizeof(double)));
      x=(double*)malloc(nblk*sizeof(double));
      CHECK(hipDeviceSynchronize());
      iStart = seconds();
      reduceUnrolling<<<grid2, block2>>>(d_v, d_x, npts);
      CHECK(hipDeviceSynchronize());
      iElaps = seconds() - iStart;
      CHECK(hipMemcpy(x, d_x, nblk * sizeof(double),hipMemcpyDeviceToHost));

      double gpu_sum = 0.0;
      for (int i = 0; i < grid2.x; i++) gpu_sum += x[i];

      printf("gpu Unrolling  elapsed %f sec gpu_sum: %f <<<grid %d block "
             "%d>>>\n", iElaps, gpu_sum, grid2.x, block2.x);

      assert(abs((gpu_sum-cpu_sum)/cpu_sum)<sqrt(npts)*DBL_EPSILON);
   }

   // Einheitsvektor
   memset(v, 0, nBytes);
   v[coord2index(Nx/2,Nx/2)]=1.0; // v=0, ausser am Gitterpunkt (Nx/2+1,Ny/2+1)
   print_vector("v",v,1);*/
   
   // cg auf gpu
   
    // Toleranz, Arraysize & Iterationsgrenze festlegen
   double tol = 1e-6;
   unsigned int kmax = 1e3;
   unsigned int k = 0;
   
   double size=Nx*Ny;
   
   
   // block dim grid dim
     dim3 block(dimx,dimy);
     dim3 grid(((Nx+1+block.x)/block.x), ((Ny+1+block.y)/block.y));
   
   // 0. Iteration
   
   laplace_2d_gpu<<<grid,block>>>(d_s,d_v,Nx,Ny);
   prod_gpu<<<grid,block>>>(d_vo,d_v,d_v,Nx,Ny);
   reduceUnrolling<<<grid,block>>>(d_vo,d_rnorm_alt,size);
   prod_gpu<<<grid,block>>>(d_svo,d_s,d_v,Nx,Ny);
   reduceUnrolling<<<grid,block>>>(d_svo,d_sv,size);
   
   double d_alpha= *d_rnorm_alt/ *d_sv;
   
   mul_add_gpu<<<grid,block>>>(d_x,d_alpha,d_v,Nx,Ny);
   vec_add_gpu<<<grid,block>>>(d_r,d_v,(-d_alpha),d_s,Nx,Ny);
   
   prod_gpu<<<grid,block>>>(d_ro,d_r,d_r,Nx,Ny);
   reduceUnrolling<<<grid,block>>>(d_ro,d_rnorm,size);
   
   // Iteration
   
   while (k<kmax && *rnorm>tol)
    {
      double d_beta = *d_rnorm/ *d_rnorm_alt;			// beta= rnorm/rnormalt
      update_p_gpu<<<grid,block>>>(d_r,d_beta,d_v,Nx,Ny);	// v = r+beta*v
      assign_v2v_gpu<<<grid,block>>>(d_rnorm_alt,d_rnorm,Nx,Ny);// rnormalt = rnorm
      laplace_2d_gpu<<<grid,block>>>(d_s,d_v,Nx,Ny);		// laplace (s,v)
      
      prod_gpu<<<grid,block>>>(d_vro,d_v,d_r,Nx,Ny);			// skalarprod v*r
      reduceUnrolling<<<grid,block>>>(d_vro,d_vr,size);		// skalarprod v*r
      prod_gpu<<<grid,block>>>(d_vso,d_v,d_s,Nx,Ny);			// skalarprod v*s
      reduceUnrolling<<<grid,block>>>(d_vso,d_vs,size);		// skalarprod v*s
      
      d_alpha = *d_vr/ *d_vs;					// alpha = (vr)/(vs)
      mul_add_gpu<<<grid,block>>>(d_x,d_alpha,d_v,Nx,Ny);	// x = x+alpha*v
      mul_add_gpu<<<grid,block>>>(d_r,(-d_alpha),d_s,Nx,Ny);	// r = r-alpha*s
      
      prod_gpu<<<grid,block>>>(d_ro,d_r,d_r,Nx,Ny);		// rnorm
      reduceUnrolling<<<grid,block>>>(d_ro,d_rnorm,size);	// rnorm
      
      k++;
    }
      
    CHECK(hipDeviceSynchronize());
    
    CHECK(hipMemcpy(x, d_x, nBytes, hipMemcpyDeviceToHost));
   
   printf("Anzahl Iterationen: %d \n",k);
   print_vector("x_Ergebnis",x,1);
   
   CHECK(hipFree(d_x));
   CHECK(hipFree(d_s));
   CHECK(hipFree(d_vo));
   CHECK(hipFree(d_rnorm_alt));
   CHECK(hipFree(d_svo));
   CHECK(hipFree(d_r));
   CHECK(hipFree(d_ro));
   CHECK(hipFree(d_rnorm));

   
   free(active);
   free(w);
   free(v);
   free(x);
   free(s);
   free(vo);
   free(rnormalt);
   free(svo);
   free(r);
   free(ro);
   free(rnorm);
   

   return (0);
}
